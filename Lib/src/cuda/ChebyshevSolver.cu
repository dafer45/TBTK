#include "hip/hip_runtime.h"
/* Copyright 2016 Kristofer Björnson
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file ChebyshevExpander.cu
 *
 *  @author Kristofer Björnson
 */

//Flag used to work around incompatibilities between nlohmann::json and CUDA.
//This disables code in header files that depends on nlohmann::json.
#define TBTK_DISABLE_NLOHMANN_JSON

#include "TBTK/Solver/ChebyshevExpander.h"
#include "TBTK/GPUResourceManager.h"
#include "TBTK/HALinkedList.h"
#include "TBTK/Streams.h"
#include "TBTK/TBTKMacros.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/device_ptr.h>

#include <cmath>

using namespace std;

namespace TBTK{
namespace Solver{

complex<double> minus_one(-1., 0.);
complex<double> one(1., 0.);
complex<double> two(2., 0.);
complex<double> zero(0., 0.);
complex<double> i(0., 1.);

vector<complex<double>> ChebyshevExpander::calculateCoefficientsGPU(
	Index to,
	Index from
){
	vector<Index> toVector;
	toVector.push_back(to);

	return calculateCoefficientsGPU(toVector, from)[0];
}

#ifdef TBTK_CUDA_V_12
__global__
void extractCoefficients(
	complex<double>* jResult,
	complex<double>* coefficients,
	int currentCoefficient,
	int* coefficientMap,
	int numParallelCoefficients,
	int numCoefficients
){
	for (int to = blockIdx.x * blockDim.x + threadIdx.x; 
		to < numParallelCoefficients; 
		to += blockDim.x * gridDim.x)
	{
		int iterator = to*numCoefficients + currentCoefficient;
		coefficients[iterator] = jResult[coefficientMap[to]];
	}
}

vector<
	vector<std::complex<double>>
> ChebyshevExpander::calculateCoefficientsGPU(
	vector<Index> &to,
	Index from
){
	TBTKAssert(
		scaleFactor > 0,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Scale factor must be larger than zero.",
		"Use ChebyshevExpander::setScaleFactor() to set scale factor."
	);
	TBTKAssert(
		numCoefficients > 0,
		"ChebyshevExpander::calculateCoefficients()",
		"numCoefficients has to be larger than zero.",
		""
	);

	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	vector<vector<complex<double>>> coefficients;
	for(unsigned int n = 0; n < to.size(); n++){
		coefficients.push_back(vector<complex<double>>(numCoefficients,0));
	}

	const HoppingAmplitudeSet &hoppingAmplitudeSet
		= getModel().getHoppingAmplitudeSet();

	int fromBasisIndex = hoppingAmplitudeSet.getBasisIndex(from);
	thrust::host_vector<int> coefficientMap(to.size());

	for(int n = 0; n < (int)to.size(); n++){
		coefficientMap[n] =
			hoppingAmplitudeSet.getBasisIndex(to.at(n));
	}

	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "ChebyshevExpander::calculateCoefficientsGPU\n";
		Streams::out << "\tFrom Index: " << fromBasisIndex << "\n";
		Streams::out << "\tBasis size: "
			<< hoppingAmplitudeSet.getBasisSize() << "\n";
		Streams::out << "\tUsing damping: ";
		if(damping != NULL)
			Streams::out << "Yes\n";
		else
			Streams::out << "No\n";
	}

	int basisSize = hoppingAmplitudeSet.getBasisSize();
	thrust::host_vector<complex<double>> jIn1(basisSize, 0.);
	thrust::host_vector<complex<double>> jIn2(basisSize, 0.);

	//Set up initial state (|j0>)
	jIn1[fromBasisIndex] = 1.;


	for(int n = 0; n < (int)to.size(); n++)
			coefficients[0][n] = jIn1[coefficientMap[n]];

	SparseMatrix<complex<double>> sparseMatrix = hoppingAmplitudeSet.getSparseMatrix();
	sparseMatrix.setStorageFormat(SparseMatrix<complex<double>>::StorageFormat::CSR);
	const int numHoppingAmplitudes = sparseMatrix.getCSRNumMatrixElements();
	const int numRows = sparseMatrix.getNumRows();
	const unsigned int *csrRowPointers = sparseMatrix.getCSRRowPointers();
	const unsigned int *csrColumns = sparseMatrix.getCSRColumns();
	const complex<double> *csrValues = sparseMatrix.getCSRValues();

	// calculate total memory requirement for the device
	int totalMemoryRequirement
		= hoppingAmplitudeSet.getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		complex<double>
	);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		int
	);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(
		complex<double>
	);
	totalMemoryRequirement += to.size()*numCoefficients*sizeof(
		complex<double>
	);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		int
	);
	if(damping != NULL){
		totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize(
		)*sizeof(complex<double>);
	}
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA memory requirement: ";
		if(totalMemoryRequirement < 1024){
			Streams::out << totalMemoryRequirement << "B\n";
		}
		else if(totalMemoryRequirement < 1024*1024){
			Streams::out << totalMemoryRequirement/1024 << "KB\n";
		}
		else{
			Streams::out << totalMemoryRequirement/1024/1024
				<< "MB\n";
		}
	}

	//Initialize GPU
	thrust::device_vector<complex<double>> jIn1_device = jIn1; //Already allocates and copies into the device memory
	thrust::device_vector<complex<double>> jIn2_device = jIn2;
	// int *cooHARowIndices_device;
	thrust::device_vector<int> csrHARowIndices_device(csrRowPointers,
													csrRowPointers + (numRows + 1));
	thrust::device_vector<int> csrColumns_device(csrColumns, csrColumns + numHoppingAmplitudes);
	const thrust::device_vector<complex<double>> csrValues_device(csrValues, csrValues + numHoppingAmplitudes);

	//The device coefficients are stored in a flattened array as complare to the host coefficients
	thrust::device_vector<complex<double>> coefficients_device(to.size()*numCoefficients);
	thrust::device_vector<int> coefficientMap_device = coefficientMap;
	complex<double> *damping_device = NULL;

	if(damping != NULL){
		TBTKAssert(
			hipMalloc(
				(void**)&damping_device,
				hoppingAmplitudeSet.getBasisSize()*sizeof(
					complex<double>
				)
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA malloc error while allocating damping_device.",
			""
		);
	}
	//Only the zeroth coefficients have been calculated, so they are copied to the device
	for(int n = 0; n < (int)to.size(); n++){
		thrust::copy(coefficients[n].begin(),
		coefficients[n].begin() + 1,
		coefficients_device.begin() + n*numCoefficients
		);
	}

	if(damping != NULL){
		TBTKAssert(
			hipMemcpy(
				damping_device,
				damping,
				hoppingAmplitudeSet.getBasisSize()*sizeof(
					complex<double>
				),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA memcpy error while copying damping.",
			""
		);
	}

	hipsparseHandle_t handle = NULL;
	TBTKAssert(
		hipsparseCreate(&handle) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE create error.",
		""
	);

	//Create a sparse matrix on the device
    hipsparseSpMatDescr_t descr;
    TBTKAssert( hipsparseCreateCsr(&descr, numRows, numRows,
		numHoppingAmplitudes,
		csrHARowIndices_device.data().get(), 
		csrColumns_device.data().get(), 
		(void*) csrValues_device.data().get(),
					  HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					  HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F)
				== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseCreateCsr.",
		""
		);

	//Create the dense vector objects on the device
	hipsparseDnVecDescr_t vecJIn1, vecJIn2;
	TBTKAssert(
		hipsparseCreateDnVec(&vecJIn1, basisSize, jIn1_device.data().get(), HIP_C_64F)
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseCreateDnVec.",
		""
	);
	TBTKAssert(
		hipsparseCreateDnVec(&vecJIn2, basisSize, jIn2_device.data().get(), HIP_C_64F)
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseCreateDnVec.",
		""
	);



	//Calculate |j1>
	int block_size = to.size();
	int num_blocks = 1;
	int maxNumThreads = 1024;
	if(block_size > maxNumThreads){
		block_size = maxNumThreads;
		num_blocks = 1 + ((to.size() - 1) / maxNumThreads); // ceil(to.size()/maxNumThreads)
	}
	
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}


	complex<double> multiplier = one/scaleFactor;
	//Allocate buffer memory for the hipsparseSpMV routine
    void                *buffer_device = NULL;
    size_t               bufferSize = 0;
    TBTKAssert( 
		hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&multiplier, descr, vecJIn1, &zero, vecJIn2, HIP_C_64F,
			HIPSPARSE_SPMV_CSR_ALG2, &bufferSize) 
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseSpMV_bufferSize.",
		"Error occured while allocating extra buffer in device memory"
	);
    TBTKAssert(
		hipMalloc(&buffer_device, bufferSize)
		== hipSuccess,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipMalloc.",
	   "Error occured while allocating extra buffer in device memory"
   	);
	TBTKAssert(
		hipsparseSpMV(
			handle,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&multiplier, descr, vecJIn1, 
			&zero, vecJIn2,
			HIP_C_64F,
			HIPSPARSE_SPMV_CSR_ALG2, 
			buffer_device
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficentsGPU()",
		"Matrix-vector multiplication error.",
		""
	);

	size_t bufferSizeSecondOperation = 0;
    TBTKAssert( 
		hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&multiplier, descr, vecJIn1, &minus_one, vecJIn2, HIP_C_64F,
			HIPSPARSE_SPMV_CSR_ALG2, &bufferSizeSecondOperation) 
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseSpMV_bufferSize.",
		"Error occured while allocating extra buffer in device memory"
	);
	TBTKAssert(
		bufferSizeSecondOperation <= bufferSize,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in Allocating buffer for SPMV.",
		"Buffer memory requirements changed."
	);
	extractCoefficients <<< num_blocks, block_size >>> (
		jIn2_device.data().get(),
		coefficients_device.data().get(),
		1,
		coefficientMap_device.data().get(),
		to.size(),
		numCoefficients
	);
	//Switch the order of the vectors jIn1 <-> jIn2
	hipsparseDnVecDescr_t *vecJIn1_ptr = &vecJIn1;
	hipsparseDnVecDescr_t *vecJIn2_ptr = &vecJIn2;
	hipsparseDnVecDescr_t *vecJTemp_ptr = NULL;
	thrust::device_ptr<complex<double>> jIn1_device_ptr = jIn1_device.data();
	thrust::device_ptr<complex<double>> jIn2_device_ptr = jIn2_device.data();
	thrust::device_ptr<complex<double>> jTemp_device_ptr = NULL;
	
	vecJTemp_ptr = vecJIn2_ptr;
	vecJIn2_ptr = vecJIn1_ptr;
	vecJIn1_ptr = vecJTemp_ptr;

	jTemp_device_ptr = jIn2_device_ptr;
	jIn2_device_ptr = jIn1_device_ptr;
	jIn1_device_ptr = jTemp_device_ptr;

	if(getGlobalVerbose() && getVerbose())
		Streams::out << "\tProgress (100 coefficients per dot): ";

	//Iteratively calculate |jn> and corresponding Chebyshev coefficients.
	for(int n = 2; n < numCoefficients; n++){
		multiplier = two/scaleFactor;
		TBTKAssert(
			hipsparseSpMV(
				handle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE,
				&multiplier, descr, *vecJIn1_ptr, 
				&minus_one, *vecJIn2_ptr,
				HIP_C_64F,
				HIPSPARSE_SPMV_CSR_ALG2, 
				buffer_device
			) == HIPSPARSE_STATUS_SUCCESS,
			"ChebyshevExpander::calculateCoefficentsGPU()",
			"Matrix-vector multiplication error.",
			""
		);
		extractCoefficients <<< num_blocks, block_size >>> (
			jIn2_device_ptr.get(),
			coefficients_device.data().get(),
			n,
			coefficientMap_device.data().get(),
			to.size(),
			numCoefficients
		);
		vecJTemp_ptr = vecJIn2_ptr;
		vecJIn2_ptr = vecJIn1_ptr;
		vecJIn1_ptr = vecJTemp_ptr;

		jTemp_device_ptr = jIn2_device_ptr;
		jIn2_device_ptr = jIn1_device_ptr;
		jIn1_device_ptr = jTemp_device_ptr;

		if(getGlobalVerbose() && getVerbose()){
			if(n%100 == 0)
				Streams::out << "." << flush;
			if(n%1000 == 0)
				Streams::out << " " << flush;
		}
	}
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "\n";
	for(unsigned int n = 0; n < to.size(); n++){
		thrust::copy(
			coefficients_device.begin() + n*numCoefficients,
			coefficients_device.begin() + numCoefficients*(n+1),
			coefficients[n].begin()
		);
	}

    TBTKAssert(
		hipsparseDestroySpMat(descr
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroySpMat.",
	   "Error while destroying the handle descr stored on the device."	
   	)
	descr = NULL;

    TBTKAssert(
		hipsparseDestroyDnVec(vecJIn1
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroyDnVec.",
	   "Error while destroying the dense vector vecJIn1 stored on the device."	
    )
	vecJIn1 = NULL;
    TBTKAssert(
		hipsparseDestroyDnVec(vecJIn2
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroyDnVec.",
	   "Error while destroying the dense vector vecJIn2 stored on the device."	
    )
	vecJIn2 = NULL;
    TBTKAssert(
		hipFree(buffer_device
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA free error while deallocating buffer_device.",
		""
	)
	buffer_device = NULL;
    TBTKAssert(
		hipsparseDestroy(handle
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroy.",
	   "Error while destroying the handle for the cuSparse calculation."	
    )
	handle = NULL;

	if(damping != NULL)
		hipFree(damping_device);

	GPUResourceManager::getInstance().freeDevice(device);

	//Lorentzian convolution
	if(broadening != 0){
		double lambda = broadening*numCoefficients;
		for(int n = 0; n < numCoefficients; n++){
			for(int c = 0; c < (int)to.size(); c++){
				coefficients[c][n] = coefficients[c][n]*sinh(
					lambda*(
						1 - n/(double)numCoefficients
					)
				)/sinh(lambda);
//				coefficients[n + c*numCoefficients] = coefficients[n + c*numCoefficients]*sinh(lambda*(1 - n/(double)numCoefficients))/sinh(lambda);
			}
		}
	}

	return coefficients;
}
#else // TBTK_CUDA_V_12
__global__
void extractCoefficients(
	hipDoubleComplex *jResult,
	int basisSize,
	hipDoubleComplex *coefficients,
	int currentCoefficient,
	int *coefficientMap,
	int numCoefficients
){
	int to = blockIdx.x*blockDim.x + threadIdx.x;
	if(to < basisSize && coefficientMap[to] != -1){
		coefficients[
			coefficientMap[to]*numCoefficients + currentCoefficient
		] = jResult[to];
	}
}

vector<
	vector<std::complex<double>>
> ChebyshevExpander::calculateCoefficientsGPU(
	vector<Index> &to,
	Index from
){
	TBTKAssert(
		scaleFactor > 0,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Scale factor must be larger than zero.",
		"Use ChebyshevExpander::setScaleFactor() to set scale factor."
	);
	TBTKAssert(
		numCoefficients > 0,
		"ChebyshevExpander::calculateCoefficients()",
		"numCoefficients has to be larger than zero.",
		""
	);

	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	vector<vector<complex<double>>> coefficients;
	for(unsigned int n = 0; n < to.size(); n++){
		coefficients.push_back(vector<complex<double>>());
		coefficients[n].reserve(numCoefficients);
		for(int c = 0; c < numCoefficients; c++)
			coefficients[n].push_back(0);
	}

	const HoppingAmplitudeSet &hoppingAmplitudeSet
		= getModel().getHoppingAmplitudeSet();

	int fromBasisIndex = hoppingAmplitudeSet.getBasisIndex(from);
	int *coefficientMap = new int[hoppingAmplitudeSet.getBasisSize()];
	for(int n = 0; n < hoppingAmplitudeSet.getBasisSize(); n++)
		coefficientMap[n] = -1;
	for(int n = 0; n < (int)to.size(); n++){
		coefficientMap[
			hoppingAmplitudeSet.getBasisIndex(to.at(n))
		] = n;
	}

	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "ChebyshevExpander::calculateCoefficientsGPU\n";
		Streams::out << "\tFrom Index: " << fromBasisIndex << "\n";
		Streams::out << "\tBasis size: "
			<< hoppingAmplitudeSet.getBasisSize() << "\n";
		Streams::out << "\tUsing damping: ";
		if(damping != NULL)
			Streams::out << "Yes\n";
		else
			Streams::out << "No\n";
	}

	complex<double> *jIn1
		= new complex<double>[hoppingAmplitudeSet.getBasisSize()];
	complex<double> *jIn2
		= new complex<double>[hoppingAmplitudeSet.getBasisSize()];
	complex<double> *jTemp = NULL;
	for(int n = 0; n < hoppingAmplitudeSet.getBasisSize(); n++){
		jIn1[n] = 0.;
		jIn2[n] = 0.;
	}

	//Set up initial state (|j0>)
	jIn1[fromBasisIndex] = 1.;

	for(int n = 0; n < hoppingAmplitudeSet.getBasisSize(); n++)
		if(coefficientMap[n] != -1)
			coefficients[coefficientMap[n]][0] = jIn1[n];
//			coefficients[coefficientMap[n]*numCoefficients] = jIn1[n];

	SparseMatrix<complex<double>> sparseMatrix = hoppingAmplitudeSet.getSparseMatrix();
	sparseMatrix.setStorageFormat(SparseMatrix<complex<double>>::StorageFormat::CSR);

	const int numHoppingAmplitudes = sparseMatrix.getCSRNumMatrixElements();
	const unsigned int *csrRowPointers = sparseMatrix.getCSRRowPointers();
	const unsigned int *csrColumns = sparseMatrix.getCSRColumns();
	const complex<double> *csrValues = sparseMatrix.getCSRValues();
	int *cooHARowIndices_host = new int[numHoppingAmplitudes];
	int *cooHAColIndices_host = new int[numHoppingAmplitudes];
	complex<double> *cooHAValues_host = new complex<double>[
		numHoppingAmplitudes
	];
	for(
		unsigned int row = 0;
		row < sparseMatrix.getNumRows();
		row++
	){
		for(
			unsigned int n = csrRowPointers[row];
			n < csrRowPointers[row+1];
			n++
		){
			cooHARowIndices_host[n] = row;
			cooHAColIndices_host[n] = csrColumns[n];
			cooHAValues_host[n] = csrValues[n];
		}
	}

	//Initialize GPU
	complex<double> *jIn1_device;
	complex<double> *jIn2_device;
	int *cooHARowIndices_device;
	int *csrHARowIndices_device;
	int *cooHAColIndices_device;
	complex<double> *cooHAValues_device;
	complex<double> *coefficients_device;
	int *coefficientMap_device;
	complex<double> *damping_device = NULL;

	int totalMemoryRequirement
		= hoppingAmplitudeSet.getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		complex<double>
	);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		int
	);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(
		complex<double>
	);
	totalMemoryRequirement += to.size()*numCoefficients*sizeof(
		complex<double>
	);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		int
	);
	if(damping != NULL){
		totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize(
		)*sizeof(complex<double>);
	}
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA memory requirement: ";
		if(totalMemoryRequirement < 1024){
			Streams::out << totalMemoryRequirement/1024 << "B\n";
		}
		else if(totalMemoryRequirement < 1024*1024){
			Streams::out << totalMemoryRequirement/1024 << "KB\n";
		}
		else{
			Streams::out << totalMemoryRequirement/1024/1024
				<< "MB\n";
		}
	}

	TBTKAssert(
		hipMalloc(
			(void**)&jIn1_device,
			hoppingAmplitudeSet.getBasisSize()*sizeof(
				complex<double>
			)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating jIn1_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&jIn2_device,
			hoppingAmplitudeSet.getBasisSize()*sizeof(
				complex<double>
			)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating jIn2_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&cooHARowIndices_device,
			numHoppingAmplitudes*sizeof(int)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating cooHARowIndices_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&csrHARowIndices_device,
			(hoppingAmplitudeSet.getBasisSize()+1)*sizeof(int)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating csrHARowIndices_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&cooHAColIndices_device,
			numHoppingAmplitudes*sizeof(int)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating cooHAColIndices_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&cooHAValues_device,
			numHoppingAmplitudes*sizeof(complex<double>)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating cooHAValues_device.",
		""
	)
	TBTKAssert(
		hipMalloc(
			(void**)&coefficients_device,
			to.size()*numCoefficients*sizeof(complex<double>)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating coefficients_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&coefficientMap_device,
			hoppingAmplitudeSet.getBasisSize()*sizeof(int)
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA malloc error while allocating coefficientMap_device.",
		""
	);
	if(damping != NULL){
		TBTKAssert(
			hipMalloc(
				(void**)&damping_device,
				hoppingAmplitudeSet.getBasisSize()*sizeof(
					complex<double>
				)
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA malloc error while allocating damping_device.",
			""
		);
	}

	TBTKAssert(
		hipMemcpy(
			jIn1_device,
			jIn1,
			hoppingAmplitudeSet.getBasisSize()*sizeof(
				complex<double>
			),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying jIn1.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			jIn2_device,
			jIn2,
			hoppingAmplitudeSet.getBasisSize()*sizeof(
				complex<double>
			),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying jIn2.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			cooHARowIndices_device,
			cooHARowIndices_host,
			numHoppingAmplitudes*sizeof(int),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying cooHARowIndices.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			cooHAColIndices_device,
			cooHAColIndices_host,
			numHoppingAmplitudes*sizeof(int),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficients()",
		"CUDA memcpy error while copying cooHAColIndices.",
		""
	)
	TBTKAssert(
		hipMemcpy(
			cooHAValues_device,
			cooHAValues_host,
			numHoppingAmplitudes*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying cooHAValues.",
		""
	);
	for(unsigned int n = 0; n < to.size(); n++){
		TBTKAssert(
			hipMemcpy(
				coefficients_device + numCoefficients*n,
				coefficients[n].data(),
				numCoefficients*sizeof(complex<double>),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficients()",
			"CUDA memcpy error while copying coefficients.",
			""
		);
	}
/*	TBTKAssert(
		hipMemcpy(
			coefficients_device,
			coefficients.data(),
			to.size()*numCoefficients*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficients()",
		"CUDA memcpy error while copying coefficients.",
		""
	);*/
	TBTKAssert(
		hipMemcpy(
			coefficientMap_device,
			coefficientMap,
			hoppingAmplitudeSet.getBasisSize()*sizeof(int),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying coefficientMap.",
		""
	);
	if(damping != NULL){
		TBTKAssert(
			hipMemcpy(
				damping_device,
				damping,
				hoppingAmplitudeSet.getBasisSize()*sizeof(
					complex<double>
				),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA memcpy error while copying damping.",
			""
		);
	}

	hipsparseHandle_t handle = NULL;
	TBTKAssert(
		hipsparseCreate(&handle) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE create error.",
		""
	);

	hipsparseMatDescr_t descr = NULL;
	TBTKAssert(
		hipsparseCreateMatDescr(&descr) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE create matrix descriptor error.",
		""
	);

	TBTKAssert(
		hipsparseSetMatType(
			descr,
			HIPSPARSE_MATRIX_TYPE_GENERAL
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE set matrix type error.",
		""
	);
	TBTKAssert(
		hipsparseSetMatIndexBase(
			descr,
			HIPSPARSE_INDEX_BASE_ZERO
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE set matrix index base error.",
		""
	);

	TBTKAssert(
		hipsparseXcoo2csr(
			handle,
			cooHARowIndices_device,
			numHoppingAmplitudes,
			hoppingAmplitudeSet.getBasisSize(),
			csrHARowIndices_device,
			HIPSPARSE_INDEX_BASE_ZERO
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE COO to CSR error.",
		""
	);

	//Calculate |j1>
	int block_size = 1024;
	int num_blocks = hoppingAmplitudeSet.getBasisSize()/block_size
		+ (hoppingAmplitudeSet.getBasisSize()%block_size == 0 ? 0:1);
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}

	complex<double> multiplier = one/scaleFactor;
	TBTKAssert(
		hipsparseZcsrmv(
			handle,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			hoppingAmplitudeSet.getBasisSize(),
			hoppingAmplitudeSet.getBasisSize(),
			numHoppingAmplitudes,
			(hipDoubleComplex*)&multiplier,
			descr,
			(hipDoubleComplex*)cooHAValues_device,
			csrHARowIndices_device,
			cooHAColIndices_device,
			(hipDoubleComplex*)jIn1_device,
			(hipDoubleComplex*)&zero,
			(hipDoubleComplex*)jIn2_device
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficentsGPU()",
		"Matrix-vector multiplication error.",
		""
	);

	extractCoefficients <<< num_blocks, block_size >>> (
		(hipDoubleComplex*)jIn2_device,
		hoppingAmplitudeSet.getBasisSize(),
		(hipDoubleComplex*)coefficients_device,
		1,
		coefficientMap_device,
		numCoefficients
	);
	jTemp = jIn2_device;
	jIn2_device = jIn1_device;
	jIn1_device = jTemp;

	if(getGlobalVerbose() && getVerbose())
		Streams::out << "\tProgress (100 coefficients per dot): ";

	//Iteratively calculate |jn> and corresponding Chebyshev coefficients.
	for(int n = 2; n < numCoefficients; n++){
		multiplier = two/scaleFactor;
		TBTKAssert(
			hipsparseZcsrmv(
				handle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE,
				hoppingAmplitudeSet.getBasisSize(),
				hoppingAmplitudeSet.getBasisSize(),
				numHoppingAmplitudes,
				(hipDoubleComplex*)&multiplier,
				descr,
				(hipDoubleComplex*)cooHAValues_device,
				csrHARowIndices_device,
				cooHAColIndices_device,
				(hipDoubleComplex*)jIn1_device,
				(hipDoubleComplex*)&minus_one,
				(hipDoubleComplex*)jIn2_device
			) == HIPSPARSE_STATUS_SUCCESS,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"Matrix-vector multiplication error.",
			""
		);

		extractCoefficients <<< num_blocks, block_size >>> (
			(hipDoubleComplex*)jIn2_device,
			hoppingAmplitudeSet.getBasisSize(),
			(hipDoubleComplex*)coefficients_device,
			n,
			coefficientMap_device,
			numCoefficients
		);

		jTemp = jIn2_device;
		jIn2_device = jIn1_device;
		jIn1_device = jTemp;

		if(getGlobalVerbose() && getVerbose()){
			if(n%100 == 0)
				Streams::out << "." << flush;
			if(n%1000 == 0)
				Streams::out << " " << flush;
		}
	}
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "\n";

	for(unsigned int n = 0; n < to.size(); n++){
		TBTKAssert(
			hipMemcpy(
				coefficients[n].data(),
				coefficients_device + numCoefficients*n,
				numCoefficients*sizeof(complex<double>),
				hipMemcpyDeviceToHost
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA memcpy error while copying coefficients.",
			""
		);
	}
/*	TBTKAssert(
		hipMemcpy(
			coefficients.data(),
			coefficients_device,
			to.size()*numCoefficients*sizeof(complex<double>),
			hipMemcpyDeviceToHost
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA memcpy error while copying coefficients.",
		""
	);*/

	TBTKAssert(
		hipsparseDestroyMatDescr(descr) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE destroy matrix descriptor error.",
		""
	);
	descr = NULL;

	TBTKAssert(
		hipsparseDestroy(handle) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE destroy error.",
		""
	);
	handle = NULL;

	delete [] jIn1;
	delete [] jIn2;
	delete [] coefficientMap;
	delete [] cooHARowIndices_host;
	delete [] cooHAColIndices_host;
	delete [] cooHAValues_host;

	hipFree(jIn1_device);
	hipFree(jIn2_device);
	hipFree(cooHARowIndices_device);
	hipFree(csrHARowIndices_device);
	hipFree(cooHAColIndices_device);
	hipFree(cooHAValues_device);
	hipFree(coefficients_device);
	hipFree(coefficientMap_device);
	if(damping != NULL)
		hipFree(damping_device);

	GPUResourceManager::getInstance().freeDevice(device);

	//Lorentzian convolution
	if(broadening != 0){
		double lambda = broadening*numCoefficients;
		for(int n = 0; n < numCoefficients; n++){
			for(int c = 0; c < (int)to.size(); c++){
				coefficients[c][n] = coefficients[c][n]*sinh(
					lambda*(
						1 - n/(double)numCoefficients
					)
				)/sinh(lambda);
//				coefficients[n + c*numCoefficients] = coefficients[n + c*numCoefficients]*sinh(lambda*(1 - n/(double)numCoefficients))/sinh(lambda);
			}
		}
	}

	return coefficients;
}

#endif // TBTK_CUDA_V_12

__global__
void calculateGreensFunction(
	hipDoubleComplex *greensFunction,
	hipDoubleComplex *coefficients,
	hipDoubleComplex *lookupTable,
	int numCoefficients,
	int energyResolution
){
	int e = blockIdx.x*blockDim.x + threadIdx.x;
	if(e < energyResolution){
		for(int n = 0; n < numCoefficients; n++){
			greensFunction[e] = hipCadd(
				greensFunction[e],
				hipCmul(
					lookupTable[n*energyResolution + e],
					coefficients[n]
				)
			);
		}
	}
}

void ChebyshevExpander::loadLookupTableGPU(){
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "CheyshevExpander::loadLookupTableGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable != NULL,
		"ChebyshevExpander::loadLookupTableGPU()",
		"Lookup table has not been generated.",
		"Call ChebyshevExpander::generateLokupTable() to generate"
		<< " lookup table."
	);
	if(generatingFunctionLookupTable_device != NULL)
		destroyLookupTableGPU();

	complex<double> *generatingFunctionLookupTable_host
		= new complex<double>[
			lookupTableNumCoefficients*lookupTableResolution
		];
	for(int n = 0; n < lookupTableNumCoefficients; n++){
		for(int e = 0; e < lookupTableResolution; e++){
			generatingFunctionLookupTable_host[
				n*lookupTableResolution + e
			] = generatingFunctionLookupTable[n][e];
		}
	}

	int memoryRequirement
		= lookupTableNumCoefficients*lookupTableResolution*sizeof(
			complex<double>
		);
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA memory requirement: ";
		if(memoryRequirement < 1024)
			Streams::out << memoryRequirement << "B\n";
		else if(memoryRequirement < 1024*1024)
			Streams::out << memoryRequirement/1024 << "KB\n";
		else
			Streams::out << memoryRequirement/1024/1024 << "MB\n";
	}

	generatingFunctionLookupTable_device = new complex<double>**[
		GPUResourceManager::getInstance().getNumDevices()
	];

	for(
		int n = 0;
		n < GPUResourceManager::getInstance().getNumDevices();
		n++
	){
		TBTKAssert(
			hipSetDevice(n) == hipSuccess,
			"ChebyshevExpander::loadLookupTableGPU()",
			"CUDA set device error for device " << n << ".",
			""
		);

		TBTKAssert(
			hipMalloc(
				(void**)&generatingFunctionLookupTable_device[
					n
				],
				lookupTableNumCoefficients*lookupTableResolution*sizeof(
					complex<double>
				)
			)  == hipSuccess,
			"ChebyshevExpander::loadLookupTableGPU()",
			"CUDA malloc error while allocating"
			<< " generatingFunctionLookupTable_device.",
			""
		);

		TBTKAssert(
			hipMemcpy(
				generatingFunctionLookupTable_device[n],
				generatingFunctionLookupTable_host,
				lookupTableNumCoefficients*lookupTableResolution*sizeof(
					complex<double>
				),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevExpander::loadLookupTableGPU()",
			"CUDA memcpy error while copying"
			<< " generatingFunctionLookupTable_device.",
			""
		);
	}

	delete [] generatingFunctionLookupTable_host;
}

void ChebyshevExpander::destroyLookupTableGPU(){
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "ChebyshevExpander::destroyLookupTableGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable_device != NULL,
		"ChebyshevExpander::destroyLookupTableGPU()",
		"No lookup table loaded onto GPU.\n",
		""
	);

	for(
		int n = 0;
		n < GPUResourceManager::getInstance().getNumDevices();
		n++
	){
		hipFree(generatingFunctionLookupTable_device[n]);
	}

	delete [] generatingFunctionLookupTable_device;
	generatingFunctionLookupTable_device = NULL;
}

//Property::GreensFunction* ChebyshevExpander::generateGreensFunctionGPU(
vector<complex<double>> ChebyshevExpander::generateGreensFunctionGPU(
	const vector<complex<double>> &coefficients,
//	Property::GreensFunction::Type type
	Type type
){
	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	ensureLookupTableIsReady();

	if(getGlobalVerbose() && getVerbose())
		Streams::out << "ChebyshevExpander::generateGreensFunctionGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable_device != NULL,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"No lookup table loaded onto GPU.",
		""
	);
	TBTKAssert(
//		type == Property::GreensFunction::Type::Retarded,
		type == Type::Retarded,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"Only evaluation of retarded Green's function is implemented"
		<< " for GPU so far.",
		"Use CPU evaluation instead."
	);

/*	complex<double> *greensFunctionData = new complex<double>[
		lookupTableResolution
	];

	for(int e = 0; e < lookupTableResolution; e++)
		greensFunctionData[e] = 0.;*/

	vector<complex<double>> greensFunctionData(lookupTableResolution, 0.);

	complex<double> *greensFunctionData_device;
	complex<double> *coefficients_device;

	TBTKAssert(
		hipMalloc(
			(void**)&greensFunctionData_device,
			lookupTableResolution*sizeof(complex<double>)
		)  == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA malloc error while allocating greensFunction_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&coefficients_device,
			lookupTableNumCoefficients*sizeof(complex<double>)
		)  == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA malloc error while allocating coefficients_device.",
		""
	);

	TBTKAssert(
		hipMemcpy(
			greensFunctionData_device,
			greensFunctionData.data(),
			lookupTableResolution*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying greensFunctionData.",
		""
	);
	TBTKAssert(
		hipMemcpy(
			coefficients_device,
			coefficients.data(),
			lookupTableNumCoefficients*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying coefficients.",
		""
	);

	int block_size = 1024;
	int num_blocks = lookupTableResolution/block_size
		+ (lookupTableResolution%block_size == 0 ? 0:1);

	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}

	calculateGreensFunction <<< num_blocks, block_size>>> (
		(hipDoubleComplex*)greensFunctionData_device,
		(hipDoubleComplex*)coefficients_device,
		(hipDoubleComplex*)generatingFunctionLookupTable_device[device],
		lookupTableNumCoefficients,
		lookupTableResolution
	);

	TBTKAssert(
		hipMemcpy(
			greensFunctionData.data(),
			greensFunctionData_device,
			lookupTableResolution*sizeof(complex<double>),
			hipMemcpyDeviceToHost
		) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying greensFunction_device.",
		""
	);

	hipFree(greensFunctionData_device);
	hipFree(coefficients_device);

	GPUResourceManager::getInstance().freeDevice(device);

/*	Property::GreensFunction *greensFunction = new Property::GreensFunction(
		type,
//		Property::GreensFunction::Format::Array,
		lookupTableLowerBound,
		lookupTableUpperBound,
		lookupTableResolution,
		greensFunctionData
	);
	delete [] greensFunctionData;

	return greensFunction;*/

	return greensFunctionData;
}

};	//End of namespace Solver
};	//End of namespace TBTK
