#include "hip/hip_runtime.h"
/* Copyright 2016 Kristofer Björnson
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file ChebyshevExpander.cu
 *
 *  @author Kristofer Björnson
 */

//Flag used to work around incompatibilities between nlohmann::json and CUDA.
//This disables code in header files that depends on nlohmann::json.
// #define TBTK_DISABLE_NLOHMANN_JSON //TODO compiles fine without the flag?

#include "TBTK/Solver/ChebyshevExpander.h"
#include "TBTK/GPUResourceManager.h"
#include "TBTK/HALinkedList.h"
#include "TBTK/Streams.h"
#include "TBTK/TBTKMacros.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/device_ptr.h>

#include <cmath>

using namespace std;

namespace TBTK{
namespace Solver{

complex<double> minus_one(-1., 0.);
complex<double> one(1., 0.);
complex<double> two(2., 0.);
complex<double> zero(0., 0.);
complex<double> i(0., 1.);




__global__
void extractCoefficients(
	complex<double>* jResult,
	complex<double>* coefficients,
	int currentCoefficient,
	int* coefficientMap,
	int numParallelCoefficients,
	int numCoefficients
){
	for (int to = blockIdx.x * blockDim.x + threadIdx.x; 
		to < numParallelCoefficients; 
		to += blockDim.x * gridDim.x)
	{
		int iterator = to*numCoefficients + currentCoefficient;
		coefficients[iterator] = jResult[coefficientMap[to]];
	}
}



vector<complex<double>> ChebyshevExpander::calculateCoefficientsGPU(
	Index to,
	Index from
){
	vector<Index> toVector;
	toVector.push_back(to);

	return calculateCoefficientsGPU(toVector, from)[0];
}

vector<
	vector<std::complex<double>>
> ChebyshevExpander::calculateCoefficientsGPU(
	vector<Index> &to,
	Index from
){
	complex<double> *damping = NULL; //TODO Compiler error of damping not defined without this, note that it disables some parts of the code?
	TBTKAssert(
		scaleFactor > 0,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Scale factor must be larger than zero.",
		"Use ChebyshevExpander::setScaleFactor() to set scale factor."
	);
	TBTKAssert(
		numCoefficients > 0,
		"ChebyshevExpander::calculateCoefficients()",
		"numCoefficients has to be larger than zero.",
		""
	);

	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	vector<vector<complex<double>>> coefficients;
	for(unsigned int n = 0; n < to.size(); n++){
		coefficients.push_back(vector<complex<double>>(numCoefficients,0));
	}

	const HoppingAmplitudeSet &hoppingAmplitudeSet
		= getModel().getHoppingAmplitudeSet();

	int fromBasisIndex = hoppingAmplitudeSet.getBasisIndex(from);
	thrust::host_vector<int> coefficientMap(to.size());

	for(int n = 0; n < (int)to.size(); n++){
		coefficientMap[n] =
			hoppingAmplitudeSet.getBasisIndex(to.at(n));
	}

	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "ChebyshevExpander::calculateCoefficientsGPU\n";
		Streams::out << "\tFrom Index: " << fromBasisIndex << "\n";
		Streams::out << "\tBasis size: "
			<< hoppingAmplitudeSet.getBasisSize() << "\n";
		Streams::out << "\tUsing damping: ";
		if(damping != NULL)
			Streams::out << "Yes\n";
		else
			Streams::out << "No\n";
	}

	int basisSize = hoppingAmplitudeSet.getBasisSize();
	thrust::host_vector<complex<double>> jIn1(basisSize, 0.);
	thrust::host_vector<complex<double>> jIn2(basisSize, 0.);

	//Set up initial state (|j0>)
	jIn1[fromBasisIndex] = 1.;


	for(int n = 0; n < (int)to.size(); n++)
			coefficients[0][n] = jIn1[coefficientMap[n]];

	SparseMatrix<complex<double>> sparseMatrix = hoppingAmplitudeSet.getSparseMatrix();
	sparseMatrix.setStorageFormat(SparseMatrix<complex<double>>::StorageFormat::CSR);
	const int numHoppingAmplitudes = sparseMatrix.getCSRNumMatrixElements();
	const int numRows = sparseMatrix.getNumRows();
	const unsigned int *csrRowPointers = sparseMatrix.getCSRRowPointers();
	const unsigned int *csrColumns = sparseMatrix.getCSRColumns();
	const complex<double> *csrValues = sparseMatrix.getCSRValues();

	// calculate total memory requirement for the device
	int totalMemoryRequirement
		= hoppingAmplitudeSet.getBasisSize()*sizeof(complex<double>);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		complex<double>
	);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		int
	);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(int);
	totalMemoryRequirement += numHoppingAmplitudes*sizeof(
		complex<double>
	);
	totalMemoryRequirement += to.size()*numCoefficients*sizeof(
		complex<double>
	);
	totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize()*sizeof(
		int
	);
	if(damping != NULL){
		totalMemoryRequirement += hoppingAmplitudeSet.getBasisSize(
		)*sizeof(complex<double>);
	}
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA memory requirement: ";
		if(totalMemoryRequirement < 1024){
			Streams::out << totalMemoryRequirement << "B\n";
		}
		else if(totalMemoryRequirement < 1024*1024){
			Streams::out << totalMemoryRequirement/1024 << "KB\n";
		}
		else{
			Streams::out << totalMemoryRequirement/1024/1024
				<< "MB\n";
		}
	}

	//Initialize GPU
	thrust::device_vector<complex<double>> jIn1_device = jIn1; //Already allocates and copies into the device memory
	thrust::device_vector<complex<double>> jIn2_device = jIn2;
	// int *cooHARowIndices_device;
	thrust::device_vector<int> csrHARowIndices_device(csrRowPointers,
													csrRowPointers + (numRows + 1));
	thrust::device_vector<int> csrColumns_device(csrColumns, csrColumns + numHoppingAmplitudes);
	const thrust::device_vector<complex<double>> csrValues_device(csrValues, csrValues + numHoppingAmplitudes);

	//The device coefficients are stored in a flattened array as complare to the host coefficients
	thrust::device_vector<complex<double>> coefficients_device(to.size()*numCoefficients);
	thrust::device_vector<int> coefficientMap_device = coefficientMap;
	complex<double> *damping_device = NULL;

	if(damping != NULL){
		TBTKAssert(
			hipMalloc(
				(void**)&damping_device,
				hoppingAmplitudeSet.getBasisSize()*sizeof(
					complex<double>
				)
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA malloc error while allocating damping_device.",
			""
		);
	}
	//Only the zeroth coefficients have been calculated, so they are copied to the device
	for(int n = 0; n < (int)to.size(); n++){
		thrust::copy(coefficients[n].begin(),
		coefficients[n].begin() + 1,
		coefficients_device.begin() + n*numCoefficients
		);
	}

	if(damping != NULL){
		TBTKAssert(
			hipMemcpy(
				damping_device,
				damping,
				hoppingAmplitudeSet.getBasisSize()*sizeof(
					complex<double>
				),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevExpander::calculateCoefficientsGPU()",
			"CUDA memcpy error while copying damping.",
			""
		);
	}

	hipsparseHandle_t handle = NULL;
	TBTKAssert(
		hipsparseCreate(&handle) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"cuSPARSE create error.",
		""
	);

	//Create a sparse matrix on the device
    hipsparseSpMatDescr_t descr;
    TBTKAssert( hipsparseCreateCsr(&descr, numRows, numRows,
		numHoppingAmplitudes,
		csrHARowIndices_device.data().get(), 
		csrColumns_device.data().get(), 
		(void*) csrValues_device.data().get(),
					  HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					  HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F)
				== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseCreateCsr.",
		""
		);

	//Create the dense vector objects on the device
	hipsparseDnVecDescr_t vecJIn1, vecJIn2;
	TBTKAssert(
		hipsparseCreateDnVec(&vecJIn1, basisSize, jIn1_device.data().get(), HIP_C_64F)
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseCreateDnVec.",
		""
	);
	TBTKAssert(
		hipsparseCreateDnVec(&vecJIn2, basisSize, jIn2_device.data().get(), HIP_C_64F)
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseCreateDnVec.",
		""
	);



	//Calculate |j1>
	int block_size = to.size();
	int num_blocks = 1;
	int maxNumThreads = 1024;
	if(block_size > maxNumThreads){
		block_size = maxNumThreads;
		num_blocks = 1 + ((to.size() - 1) / maxNumThreads); // ceil(to.size()/maxNumThreads)
	}
	
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}


	complex<double> multiplier = one/scaleFactor;
	//Allocate buffer memory for the hipsparseSpMV routine
    void                *buffer_device = NULL;
    size_t               bufferSize = 0;
    TBTKAssert( 
		hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&multiplier, descr, vecJIn1, &zero, vecJIn2, HIP_C_64F,
			HIPSPARSE_SPMV_CSR_ALG2, &bufferSize) 
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseSpMV_bufferSize.",
		"Error occured while allocating extra buffer in device memory"
	);
    TBTKAssert(
		hipMalloc(&buffer_device, bufferSize)
		== hipSuccess,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipMalloc.",
	   "Error occured while allocating extra buffer in device memory"
   	);
	TBTKAssert(
		hipsparseSpMV(
			handle,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&multiplier, descr, vecJIn1, 
			&zero, vecJIn2,
			HIP_C_64F,
			HIPSPARSE_SPMV_CSR_ALG2, 
			buffer_device
		) == HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficentsGPU()",
		"Matrix-vector multiplication error.",
		""
	);

	size_t bufferSizeSecondOperation = 0;
    TBTKAssert( 
		hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&multiplier, descr, vecJIn1, &minus_one, vecJIn2, HIP_C_64F,
			HIPSPARSE_SPMV_CSR_ALG2, &bufferSizeSecondOperation) 
		== HIPSPARSE_STATUS_SUCCESS,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in hipsparseSpMV_bufferSize.",
		"Error occured while allocating extra buffer in device memory"
	);
	TBTKAssert(
		bufferSizeSecondOperation <= bufferSize,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"Error in Allocating buffer for SPMV.",
		"Buffer memory requirements changed."
	);
	extractCoefficients <<< num_blocks, block_size >>> (
		jIn2_device.data().get(),
		coefficients_device.data().get(),
		1,
		coefficientMap_device.data().get(),
		to.size(),
		numCoefficients
	);
	//Switch the order of the vectors jIn1 <-> jIn2
	hipsparseDnVecDescr_t *vecJIn1_ptr = &vecJIn1;
	hipsparseDnVecDescr_t *vecJIn2_ptr = &vecJIn2;
	hipsparseDnVecDescr_t *vecJTemp_ptr = NULL;
	thrust::device_ptr<complex<double>> jIn1_device_ptr = jIn1_device.data();
	thrust::device_ptr<complex<double>> jIn2_device_ptr = jIn2_device.data();
	thrust::device_ptr<complex<double>> jTemp_device_ptr = NULL;
	
	vecJTemp_ptr = vecJIn2_ptr;
	vecJIn2_ptr = vecJIn1_ptr;
	vecJIn1_ptr = vecJTemp_ptr;

	jTemp_device_ptr = jIn2_device_ptr;
	jIn2_device_ptr = jIn1_device_ptr;
	jIn1_device_ptr = jTemp_device_ptr;

	if(getGlobalVerbose() && getVerbose())
		Streams::out << "\tProgress (100 coefficients per dot): ";

	//Iteratively calculate |jn> and corresponding Chebyshev coefficients.
	for(int n = 2; n < numCoefficients; n++){
		multiplier = two/scaleFactor;
		TBTKAssert(
			hipsparseSpMV(
				handle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE,
				&multiplier, descr, *vecJIn1_ptr, 
				&minus_one, *vecJIn2_ptr,
				HIP_C_64F,
				HIPSPARSE_SPMV_CSR_ALG2, 
				buffer_device
			) == HIPSPARSE_STATUS_SUCCESS,
			"ChebyshevExpander::calculateCoefficentsGPU()",
			"Matrix-vector multiplication error.",
			""
		);
		extractCoefficients <<< num_blocks, block_size >>> (
			jIn2_device_ptr.get(),
			coefficients_device.data().get(),
			n,
			coefficientMap_device.data().get(),
			to.size(),
			numCoefficients
		);
		vecJTemp_ptr = vecJIn2_ptr;
		vecJIn2_ptr = vecJIn1_ptr;
		vecJIn1_ptr = vecJTemp_ptr;

		jTemp_device_ptr = jIn2_device_ptr;
		jIn2_device_ptr = jIn1_device_ptr;
		jIn1_device_ptr = jTemp_device_ptr;

		if(getGlobalVerbose() && getVerbose()){
			if(n%100 == 0)
				Streams::out << "." << flush;
			if(n%1000 == 0)
				Streams::out << " " << flush;
		}
	}
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "\n";
	for(unsigned int n = 0; n < to.size(); n++){
		thrust::copy(
			coefficients_device.begin() + n*numCoefficients,
			coefficients_device.begin() + numCoefficients*(n+1),
			coefficients[n].begin()
		);
	}

    TBTKAssert(
		hipsparseDestroySpMat(descr
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroySpMat.",
	   "Error while destroying the handle descr stored on the device."	
   	)
	descr = NULL;

    TBTKAssert(
		hipsparseDestroyDnVec(vecJIn1
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroyDnVec.",
	   "Error while destroying the dense vector vecJIn1 stored on the device."	
    )
	vecJIn1 = NULL;
    TBTKAssert(
		hipsparseDestroyDnVec(vecJIn2
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroyDnVec.",
	   "Error while destroying the dense vector vecJIn2 stored on the device."	
    )
	vecJIn2 = NULL;
    TBTKAssert(
		hipFree(buffer_device
		) == hipSuccess,
		"ChebyshevExpander::calculateCoefficientsGPU()",
		"CUDA free error while deallocating buffer_device.",
		""
	)
	buffer_device = NULL;
    TBTKAssert(
		hipsparseDestroy(handle
	   ) == HIPSPARSE_STATUS_SUCCESS,
	   "ChebyshevExpander::calculateCoefficientsGPU()",
	   "Error in hipsparseDestroy.",
	   "Error while destroying the handle for the cuSparse calculation."	
    )
	handle = NULL;

	if(damping != NULL)
		hipFree(damping_device);

	//Lorentzian convolution
	if(broadening != 0){
		double lambda = broadening*numCoefficients;
		for(int n = 0; n < numCoefficients; n++){
			for(int c = 0; c < (int)to.size(); c++){
				coefficients[c][n] = coefficients[c][n]*sinh(
					lambda*(
						1 - n/(double)numCoefficients
					)
				)/sinh(lambda);
//				coefficients[n + c*numCoefficients] = coefficients[n + c*numCoefficients]*sinh(lambda*(1 - n/(double)numCoefficients))/sinh(lambda);
				//TODO it looks like the code is different here for the cpu version
			}
		}
	}
	GPUResourceManager::getInstance().freeDevice(device);
	return coefficients;
}

__global__
void calculateGreensFunction(
	hipDoubleComplex *greensFunction,
	hipDoubleComplex *coefficients,
	hipDoubleComplex *lookupTable,
	int numCoefficients,
	int energyResolution
){
	int e = blockIdx.x*blockDim.x + threadIdx.x;
	if(e < energyResolution){
		for(int n = 0; n < numCoefficients; n++){
			greensFunction[e] = hipCadd(
				greensFunction[e],
				hipCmul(
					lookupTable[n*energyResolution + e],
					coefficients[n]
				)
			);
		}
	}
}

void ChebyshevExpander::loadLookupTableGPU(){
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "CheyshevExpander::loadLookupTableGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable != NULL,
		"ChebyshevExpander::loadLookupTableGPU()",
		"Lookup table has not been generated.",
		"Call ChebyshevExpander::generateLokupTable() to generate"
		<< " lookup table."
	);
	if(generatingFunctionLookupTable_device != NULL)
		destroyLookupTableGPU();

	complex<double> *generatingFunctionLookupTable_host
		= new complex<double>[
			lookupTableNumCoefficients*lookupTableResolution
		];
	for(int n = 0; n < lookupTableNumCoefficients; n++){
		for(int e = 0; e < lookupTableResolution; e++){
			generatingFunctionLookupTable_host[
				n*lookupTableResolution + e
			] = generatingFunctionLookupTable[n][e];
		}
	}

	int memoryRequirement
		= lookupTableNumCoefficients*lookupTableResolution*sizeof(
			complex<double>
		);
	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA memory requirement: ";
		if(memoryRequirement < 1024)
			Streams::out << memoryRequirement << "B\n";
		else if(memoryRequirement < 1024*1024)
			Streams::out << memoryRequirement/1024 << "KB\n";
		else
			Streams::out << memoryRequirement/1024/1024 << "MB\n";
	}

	generatingFunctionLookupTable_device = new complex<double>**[
		GPUResourceManager::getInstance().getNumDevices()
	];

	for(
		int n = 0;
		n < GPUResourceManager::getInstance().getNumDevices();
		n++
	){
		TBTKAssert(
			hipSetDevice(n) == hipSuccess,
			"ChebyshevExpander::loadLookupTableGPU()",
			"CUDA set device error for device " << n << ".",
			""
		);

		TBTKAssert(
			hipMalloc(
				(void**)&generatingFunctionLookupTable_device[
					n
				],
				lookupTableNumCoefficients*lookupTableResolution*sizeof(
					complex<double>
				)
			)  == hipSuccess,
			"ChebyshevExpander::loadLookupTableGPU()",
			"CUDA malloc error while allocating"
			<< " generatingFunctionLookupTable_device.",
			""
		);

		TBTKAssert(
			hipMemcpy(
				generatingFunctionLookupTable_device[n],
				generatingFunctionLookupTable_host,
				lookupTableNumCoefficients*lookupTableResolution*sizeof(
					complex<double>
				),
				hipMemcpyHostToDevice
			) == hipSuccess,
			"ChebyshevExpander::loadLookupTableGPU()",
			"CUDA memcpy error while copying"
			<< " generatingFunctionLookupTable_device.",
			""
		);
	}

	delete [] generatingFunctionLookupTable_host;
}

void ChebyshevExpander::destroyLookupTableGPU(){
	if(getGlobalVerbose() && getVerbose())
		Streams::out << "ChebyshevExpander::destroyLookupTableGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable_device != NULL,
		"ChebyshevExpander::destroyLookupTableGPU()",
		"No lookup table loaded onto GPU.\n",
		""
	);

	for(
		int n = 0;
		n < GPUResourceManager::getInstance().getNumDevices();
		n++
	){
		hipFree(generatingFunctionLookupTable_device[n]);
	}

	delete [] generatingFunctionLookupTable_device;
	generatingFunctionLookupTable_device = NULL;
}

//Property::GreensFunction* ChebyshevExpander::generateGreensFunctionGPU(
vector<complex<double>> ChebyshevExpander::generateGreensFunctionGPU(
	const vector<complex<double>> &coefficients,
//	Property::GreensFunction::Type type
	Type type
){
	int device = GPUResourceManager::getInstance().allocateDevice();

	TBTKAssert(
		hipSetDevice(device) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA set device error for device " << device << ".",
		""
	);

	ensureLookupTableIsReady();

	if(getGlobalVerbose() && getVerbose())
		Streams::out << "ChebyshevExpander::generateGreensFunctionGPU\n";

	TBTKAssert(
		generatingFunctionLookupTable_device != NULL,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"No lookup table loaded onto GPU.",
		""
	);
	TBTKAssert(
//		type == Property::GreensFunction::Type::Retarded,
		type == Type::Retarded,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"Only evaluation of retarded Green's function is implemented"
		<< " for GPU so far.",
		"Use CPU evaluation instead."
	);

/*	complex<double> *greensFunctionData = new complex<double>[
		lookupTableResolution
	];

	for(int e = 0; e < lookupTableResolution; e++)
		greensFunctionData[e] = 0.;*/

	vector<complex<double>> greensFunctionData(lookupTableResolution, 0.);

	complex<double> *greensFunctionData_device;
	complex<double> *coefficients_device;

	TBTKAssert(
		hipMalloc(
			(void**)&greensFunctionData_device,
			lookupTableResolution*sizeof(complex<double>)
		)  == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA malloc error while allocating greensFunction_device.",
		""
	);
	TBTKAssert(
		hipMalloc(
			(void**)&coefficients_device,
			lookupTableNumCoefficients*sizeof(complex<double>)
		)  == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA malloc error while allocating coefficients_device.",
		""
	);

	TBTKAssert(
		hipMemcpyAsync(
			greensFunctionData_device,
			greensFunctionData.data(),
			lookupTableResolution*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying greensFunctionData.",
		""
	);
	TBTKAssert(
		hipMemcpyAsync(
			coefficients_device,
			coefficients.data(),
			lookupTableNumCoefficients*sizeof(complex<double>),
			hipMemcpyHostToDevice
		) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying coefficients.",
		""
	);

	int block_size = 1024;
	int num_blocks = lookupTableResolution/block_size
		+ (lookupTableResolution%block_size == 0 ? 0:1);

	if(getGlobalVerbose() && getVerbose()){
		Streams::out << "\tCUDA Block size: " << block_size << "\n";
		Streams::out << "\tCUDA Num blocks: " << num_blocks << "\n";
	}
	hipDeviceSynchronize();
	calculateGreensFunction <<< num_blocks, block_size>>> (
		(hipDoubleComplex*)greensFunctionData_device,
		(hipDoubleComplex*)coefficients_device,
		(hipDoubleComplex*)generatingFunctionLookupTable_device[device],
		lookupTableNumCoefficients,
		lookupTableResolution
	);

	TBTKAssert(
		hipMemcpy(
			greensFunctionData.data(),
			greensFunctionData_device,
			lookupTableResolution*sizeof(complex<double>),
			hipMemcpyDeviceToHost
		) == hipSuccess,
		"ChebyshevExpander::generateGreensFunctionGPU()",
		"CUDA memcpy error while copying greensFunction_device.",
		""
	);
	hipFree(greensFunctionData_device);
	hipFree(coefficients_device);
	GPUResourceManager::getInstance().freeDevice(device);

/*	Property::GreensFunction *greensFunction = new Property::GreensFunction(
		type,
//		Property::GreensFunction::Format::Array,
		lookupTableLowerBound,
		lookupTableUpperBound,
		lookupTableResolution,
		greensFunctionData
	);
	delete [] greensFunctionData;

	return greensFunction;*/

	return greensFunctionData;
}

};	//End of namespace Solver
};	//End of namespace TBTK
