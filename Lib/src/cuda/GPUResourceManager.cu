/* Copyright 2016 Kristofer Björnson
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file GPUResourceManager.cu
 *  
 *  @author Kristofer Björnson
 *  @author Andreas Theiler	
 */

#include "TBTK/GPUResourceManager.h"
#include "TBTK/Streams.h"
#include <hip/hip_runtime.h>

using namespace std;

namespace TBTK{

void GPUResourceManager::createDeviceTable(){
	hipGetDeviceCount(&numDevices);

	if(getGlobalVerbose())
		Streams::out << "Num GPU devices: " << numDevices << "\n";

	if(numDevices > 0){
		busyDevices = new bool[numDevices];
		for(int n = 0; n < numDevices; n++)
			busyDevices[n] = false;
	}
}

void GPUResourceManager::destroyDeviceTable(){
	if(numDevices > 0){
		delete [] busyDevices;
		busyDevices = NULL;
	}
}

// Code based on enablePeerAccess from Nvidias CUDA sample library
void GPUResourceManager::enableP2PAccess() {
    
	// Check if any devices are allocated
	if(!busyDevices){
		//TODO error message
		return;
	}
	hipError_t hipError_t = hipSuccess;
	int currentDevice;
	for(int i = 0; i < numDevices; i++){
		if(busyDevices[i]){
			currentDevice = i;
			// TODO error check hipSuccess // TODO not sure if this check is needed
			hipError_t = hipGetDevice(&currentDevice);
		}
	}    

    /* Remark: access granted by this hipDeviceEnablePeerAccess is unidirectional */
    /* Rows and columns represents a connectivity matrix between GPUs in the system */
    for (int activeDevice = 0; activeDevice < numDevices; activeDevice++) {
		if(busyDevices[activeDevice]){
			// TODO error check
			hipError_t = hipSetDevice(activeDevice);
		}
        for (int peer = 0; peer < numDevices; peer++) {
            if (activeDevice != peer && busyDevices[peer]) {
                int canAccessPeer = 0;
				// TODO error check
				hipError_t = hipDeviceCanAccessPeer(&canAccessPeer, activeDevice, peer);
                if (canAccessPeer) {
					// TODO error check
					hipError_t = hipDeviceEnablePeerAccess(peer, 0);
					if(getVerbose()){
						Streams::out << "P2P enabled between device " << 
										activeDevice << " and " <<
										peer <<	endl;
					}
                }
            }
        }
    }
	// TODO error check
	hipError_t = hipSetDevice(currentDevice);
}

};	//End of namespace TBTK
